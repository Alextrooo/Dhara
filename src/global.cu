#include "hip/hip_runtime.h"
/*
// Copyright (C) 2016, HydroComplexity Group
// All rights reserved.
//
// Distributed Hydrologicc and Regional Analysis (DHARA) Model
// DHARA model is free software; you can redistribute it and/or modify it under
// the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation; either version 2.1 of the License, or
// (at your option) any later version.
//
// DHARA model is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU General Public License
// for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with the software; if not, see <http://www.gnu.org/licenses/>.
//
// Author: levuvietphong@gmail.com (Phong Le)
*/

#include "../include/main.h"
#include "../include/cusplib.h"
#include "../include/devconst.h"


__device__
double maxcompglob (double a, double b)
{
    return (a < b) ? b : a;
}

__device__
double mincompglob (double a, double b) {
    return (a > b) ? b : a;
}

/**
 * @brief      Inverse Genuchten conversion of soil moisture - pressure
 *
 * @param      theta  Soil moisture [-]
 * @param      psi    Pressure head [L]
 * @param[in]  size   Size of the domain
 */
__global__ void vanGenuchtenInverse(double *theta, double *psi, int size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double lambda, m;

    while ( i < size )
    {
        lambda = n - 1.0;
        m = lambda/n;

        if (theta[i] < theta_S)
            psi[i] = -(1/alpha) * pow(pow((theta_S-theta_R)/(theta[i]-theta_R), 1/m) - 1.0, 1/n)
                                * 0.01; // [m]
        else
            psi[i] = 0;

        // Update threads if vector is long
        i += blockDim.x * gridDim.x;

    }
}



/**
 * @brief       Genuchten conversion of soil moisture - pressure
 *
 * @param      C      Specific soil moisture capacity [1/L]
 * @param      theta  Soil moisture [-]
 * @param      Ksat   Saturated hydraulic conductivity in soil [L/T]
 * @param      K      Hydraulic conductivity in soil at moisture theta [L/T]
 * @param      psi    Pressure head [L]
 * @param[in]  size   Size of the domain
 */
__global__
void vanGenuchten(double *C, double *theta, double *Ksat, double *K, double *psi, int3 globsize)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int sizexyz = globsize.x * globsize.y * globsize.z;
    double Se, _theta, _psi, lambda, m;

    while ( i < sizexyz )
    {
        lambda = n - 1.0;
        m = lambda/n;

        // Compute the volumetric moisture content [eqn 21]
        _psi = psi[i] * 100;
        if ( _psi < 0 )
            _theta = (theta_S - theta_R) / pow(1.0 + pow((alpha*(-_psi)),n), m) + theta_R;
        else
            _theta = theta_S;

        theta[i] = _theta;

        // Compute the effective saturation [eqn 2]
        Se = (_theta - theta_R)/(theta_S - theta_R);

        /* . . .Compute the hydraulic conductivity [eqn 8] . . .*/
        K[i] = Ksat[i] * sqrt(Se) * (1.0 - pow( 1.0-pow(Se,1.0/m), m) ) * (1.0 - pow( 1.0-pow( Se, 1.0/m), m ));

        // Compute the specific moisture storage derivative of eqn (21).
        // So we have to calculate C = d(theta)/dh. Then the unit is converted into [1/m].
        if (_psi < 0)
            C[i] = 100 * alpha * n * (1.0/n-1.0)*pow(alpha*abs(_psi), n-1)
                * (theta_R-theta_S) * pow(pow(alpha*abs(_psi), n)+1, 1.0/n-2.0);
        else
            C[i] = 0.0;

        // Update threads if vector is long
        i += blockDim.x * gridDim.x;

    }
}


/**
 * @brief      Set boundary conditions for 6 faces of the subsurface domain
 *
 * @param      west_bc    The west boundary condition
 * @param      east_bc    The east boundary condition
 * @param      south_bc   The south boundary condition
 * @param      north_bc   The north boundary condition
 * @param      top_bc     The top boundary condition
 * @param      bottom_bc  The bottom boundary condition
 * @param[in]  sizex      The sizeof domain in x-direction
 * @param[in]  sizey      The sizeof domain in y-direction
 * @param[in]  sizez      The sizeof domain in z-direction
 */
__global__
void SubsurfaceSetBoundaryConditionType(int *west_bc, int *east_bc, int *south_bc, int *north_bc,
                                        int *top_bc, int *bottom_bc, int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizex = globsize.x;
    int sizey = globsize.y;
    int sizez = globsize.z;

    while (tid < sizey * sizez)
    {
        west_bc[tid] = 1;
        east_bc[tid] = 1;
        __syncthreads();               // All thread must sync at this point
        tid += blockDim.x * gridDim.x; // Update threads
    }

    tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < sizex * sizez)
    {
        south_bc[tid] = 1;
        north_bc[tid] = 1;
        __syncthreads();               // All thread must sync at this point
        tid += blockDim.x * gridDim.x; // Update threads
    }

    tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < sizex * sizey)
    {
        top_bc[tid] = 1;
        bottom_bc[tid] = 1;
        __syncthreads();               // All thread must sync at this point

        tid += blockDim.x * gridDim.x; // Update threads
    }
}


/**
 * @brief      Estimate the flux entering 1st soil layer.
 *
 * @param      ph       Ponding height [L]
 * @param      hpoten   The potential pressure head [L]
 * @param      qcapa    The flux capacity of soil [L/T]
 * @param      psinp1m  Pressure head at n+1, m [L]
 * @param      knp1m    Hydraulic conductivity at n+1,m [L]
 * @param[in]  ppt      Precipitation [L]
 * @param[in]  et       Evapotranspiration [L]
 * @param[in]  sizex    Domain size in x-direction
 * @param[in]  sizey    Domain size in y-direction
 */
__global__
void EstimateFluxes(double *ph, double *hpoten, double *qcapa, double *psinp1m, double *knp1m,
                    double ppt, double et, int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizexy = globsize.x * globsize.y;

    while (tid < sizexy) {
        hpoten[tid] = ph[tid] + ppt + et;
        qcapa[tid] = -knp1m[tid]*(psinp1m[tid]-hpoten[tid]-0.5*dz) / (0.5*dz);

        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }
}



/**
 * @brief      Identify the top boundary type of subsurface flow3D model for the entire surface
 *
 * @param      hpoten    The potential pressure head [L]
 * @param      qcapa     The flux capacity of soil [L/T]
 * @param      topbc     The top boundary condition
 * @param      topqflux  The flux go through top soil layer
 * @param      psinp1m   Pressure head at n+1, m [L]
 * @param      thetan    Soil moisture at n [-]
 * @param[in]  globsize  Size of the global domain
 */
__global__
void IdentifyTopBoundary(double *hpoten, double *qcapa, int *topbc, double *topqflux,
                         double *thetan, double *ksat, int3 globsize)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int sizex = globsize.x;
    int sizey = globsize.y;

    while (tid < sizex * sizey)
    {
        topbc[tid] = 1;
        if (hpoten[tid] > 0.0)
        {
            if (hpoten[tid]/dt > qcapa[tid])
                topqflux[tid] = mincompglob(qcapa[tid], (theta_S - thetan[tid]) * dz / dt);
            else
                topqflux[tid] = mincompglob(hpoten[tid]/dt, (theta_S - thetan[tid]) * dz / dt);

            topqflux[tid] = mincompglob(topqflux[tid], ksat[tid] * dt);
        } else {
          topqflux[tid] = hpoten[tid]/dt;
        }

        // Update threads if vector is long
        tid += blockDim.x * gridDim.x;
    }
}


/**
 * @brief      Set up before running flow model
 *
 * @param      project         Class including project info
 * @param      subsurface_dev  The subsurface class in device memory
 * @param[in]  rank            Global rank of the current MPI process
 * @param[in]  procsize        Total number of MPI processes available
 * @param[in]  globsize        Size of the global domain
 * @param      cartComm        Carthesian MPI communicator
 */
void PreRunningFlowModel(ProjectClass *project, SubsurfaceFlowClass * &subsurface_dev, int rank,
                         int procsize, int3 globsize, MPI_Comm *cartComm)
{

    if (rank == MPI_MASTER_RANK)
    {
        // If any saving any data is switched on, check if output folder exist
        struct stat st = {0};
        if (stat(project->folderoutput, &st) == -1)
        {
            mkdir(project->folderoutput, 0700);
        }

        // Print out information
        printf("\n");
        printf("\nSIMULATION STARTS \n");
        printf("------------------ \n");

        // Convert pressure head (psi) to moisture (theta)
        vanGenuchten<<<TSZ,BSZ>>>(subsurface_dev->cnp1m, subsurface_dev->thetan,
                                  subsurface_dev->ksat, subsurface_dev->knp1m,
                                  subsurface_dev->psin, globsize );
        cudaCheckError("vanGenuchten");
    }

    MPI_Barrier(*cartComm);
}
